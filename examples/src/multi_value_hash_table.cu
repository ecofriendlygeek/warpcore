#include <iostream>
#include <algorithm>
#include <random>
#include "multi_value_hash_table.cuh"

int main ()
{
    using namespace warpcore;

    using key_t = std::uint64_t;
    using value_t = std::uint64_t;

    using hash_table_t = MultiValueHashTable<
        key_t,
        value_t,
        defaults::empty_key<key_t>(),
        defaults::tombstone_key<key_t>(),
        storage::multi_value::DynamicSlabListStore<value_t>,
        defaults::probing_scheme_t<key_t, 8>>;
    using status_t = typename hash_table_t::status_type;
    using status_handler_t = typename status_handlers::ReturnStatus;

    const index_t size_unique_keys = 1UL << 20;
    const index_t size_values_per_key = 4;
    const index_t size = size_unique_keys * size_values_per_key;
    const float key_load_factor = 0.95;
    const float value_load_factor = 0.4;

    const index_t key_capacity = float(size_unique_keys) / key_load_factor;
    const index_t value_capacity = float(size) / value_load_factor;

    TIMERSTART(init_table)
    hash_table_t hash_table(key_capacity, value_capacity);
    TIMERSTOP(init_table); CUERR
    std::cout << hash_table.peek_status() << std::endl;

    TIMERSTART(init_data)
    key_t * keys_unique_h = nullptr;
    hipHostMalloc(&keys_unique_h, sizeof(key_t) * size_unique_keys); CUERR
    key_t * keys_unique_d = nullptr;
    hipMalloc(&keys_unique_d, sizeof(key_t) * size_unique_keys); CUERR

    key_t * keys_in_h = nullptr;
    hipHostMalloc(&keys_in_h, sizeof(key_t) * size); CUERR
    key_t * keys_in_d = nullptr;
    hipMalloc(&keys_in_d, sizeof(key_t) * size); CUERR

    value_t * values_in_h = nullptr;
    hipHostMalloc(&values_in_h, sizeof(value_t) * size); CUERR
    value_t * values_in_d = nullptr;
    hipMalloc(&values_in_d, sizeof(value_t) * size); CUERR

    index_t * offsets_out_h = nullptr;
    hipHostMalloc(&offsets_out_h, sizeof(index_t) * size_unique_keys); CUERR
    index_t * offsets_out_d = nullptr;
    hipMalloc(&offsets_out_d, sizeof(index_t) * size_unique_keys); CUERR

    value_t * values_out_h = nullptr;
    hipHostMalloc(&values_out_h, sizeof(value_t) * size); CUERR
    value_t * values_out_d = nullptr;
    hipMalloc(&values_out_d, sizeof(value_t) * size); CUERR

    status_t * status_h = nullptr;
    hipHostMalloc(&status_h, sizeof(status_t) * size); CUERR
    status_t * status_d = nullptr;
    hipMalloc(&status_d, sizeof(status_t) * size); CUERR

    for(index_t i = 0; i < size_unique_keys; ++i)
    {
        keys_unique_h[i] = i + 1;

        for(index_t j = 0; j < size_values_per_key; ++j)
        {
            keys_in_h[i * size_values_per_key + j] = i + 1;
        }
    }

    std::random_device rd;
    std::mt19937 g(rd());

    std::shuffle(keys_in_h, keys_in_h + size, g);

    #pragma omp parallel for
    for(index_t i= 0; i < size; ++i)
    {
        values_in_h[i] = keys_in_h[i];
        status_h[i] = status_t::none();
    }

    hipMemcpy(keys_unique_d, keys_unique_h, sizeof(key_t)*size_unique_keys, H2D); CUERR
    hipMemcpy(keys_in_d, keys_in_h, sizeof(key_t)*size, H2D); CUERR
    hipMemcpy(values_in_d, values_in_h, sizeof(value_t)*size, H2D); CUERR
    hipMemset(values_out_d, 0, sizeof(value_t)*size); CUERR
    hipMemset(offsets_out_d, 0, sizeof(index_t)*size_unique_keys); CUERR
    hipMemcpy(status_d, status_h, sizeof(status_t)*size, H2D); CUERR
    TIMERSTOP(init_data); CUERR

    THROUGHPUTSTART(insert)
    hash_table.insert<status_handler_t>(
        keys_in_d,
        values_in_d,
        size,
        0,
        defaults::probing_length(),
        status_d);
    THROUGHPUTSTOP(insert, (sizeof(key_t)+sizeof(value_t)), size); CUERR
    hipDeviceSynchronize(); CUERR

    hipMemcpy(status_h, status_d, sizeof(status_t)*size, D2H); CUERR

    std::cout << "table status " << hash_table.peek_status() << std::endl;
    index_t errors = 0;
    for(index_t i = 0; i < size; ++i)
    {

        if(status_h[i].has_any())
        {
            if(errors++ < 10)
                std::cout << "STATUS: i " << i << " key " << keys_in_h[i] << " status " << status_h[i] << std::endl;
        }
    }
    if(errors >= 10)
    {
        std::cout << "...\n" << "total errors " << errors << std::endl;
    }

    std::cout << "capacity keys " << hash_table.key_capacity() << std::endl;
    std::cout << "capacity values " << hash_table.value_capacity() << std::endl;
    std::cout << "unique keys " << size_unique_keys << std::endl;
    std::cout << "values per key " << size_values_per_key << std::endl;
    std::cout << "total values " << size << std::endl;
    std::cout << "unique keys in table " << hash_table.size_keys() << std::endl;
    std::cout << "total values in table " << hash_table.size_values() << std::endl;
    std::cout << "density " << hash_table.storage_density() << std::endl;

    #pragma omp parallel for
    for(index_t i= 0; i < size; ++i)
    {
        status_h[i] = status_t::none();
    }

    hipMemcpy(status_d, status_h, sizeof(status_t)*size, H2D); CUERR

    void * temp_d = nullptr;
    index_t temp_bytes = 0;
    index_t value_size = 0;

    THROUGHPUTSTART(retrieve_dummy)
    hash_table.retrieve<status_handler_t>(
        keys_unique_d,
        size_unique_keys,
        offsets_out_d,
        values_out_d,
        value_size,
        temp_d,
        temp_bytes,
        0,
        defaults::probing_length(),
        status_d);
    THROUGHPUTSTOP(retrieve_dummy, (sizeof(key_t)+sizeof(value_t)), size); CUERR
    hipDeviceSynchronize(); CUERR

    hipMalloc(&temp_d, temp_bytes); CUERR

    THROUGHPUTSTART(retrieve)
    hash_table.retrieve<status_handler_t>(
        keys_unique_d,
        size_unique_keys,
        offsets_out_d,
        values_out_d,
        value_size,
        temp_d,
        temp_bytes,
        0,
        defaults::probing_length(),
        status_d);
    THROUGHPUTSTOP(retrieve, (sizeof(key_t)+sizeof(value_t)), size); CUERR
    hipDeviceSynchronize(); CUERR

    std::cout << "temp_bytes: " << B2KB(temp_bytes) << "KB\tvalue_size: " << value_size << std::endl;

    lambda_kernel<<<SDIV(size_unique_keys, 1024), 1024>>>([=] DEVICEQUALIFIER
    {
        const index_t tid = blockDim.x * blockIdx.x + threadIdx.x;

        if(tid < size_unique_keys)
        {
            const auto key = keys_unique_d[tid];
            const auto lower = (tid == 0) ? 0 : offsets_out_d[tid - 1];
            const auto upper = offsets_out_d[tid];

            if(upper - lower != size_values_per_key)
            {
                printf("ERROR size values %llu\n", upper - lower);
            }

            for (index_t i = lower; i < upper; i++)
            {
                if(values_out_d[i] != key)
                {
                    printf("ERROR expected %llu got %llu\n", key, values_out_d[i]);
                }
            }
        }
    });

    hipDeviceSynchronize(); CUERR

    hipMemcpy(status_h, status_d, sizeof(status_t)*size, D2H); CUERR

    std::cout << "table status " << hash_table.peek_status() << std::endl;

    errors = 0;
    for(index_t i = 0; i < size; ++i)
    {

        if(status_h[i].has_any())
        {
            if(errors++ < 10)
                std::cout << "STATUS: i " << i << " key " << keys_in_h[i] << " status " << status_h[i] << std::endl;
        }
    }
    if(errors >= 10)
    {
        std::cout << "...\n" << "total errors " << errors << std::endl;
    }

    hipHostFree(keys_unique_h);
    hipHostFree(keys_in_h);
    hipHostFree(values_in_h);
    hipHostFree(offsets_out_h);
    hipHostFree(values_out_h);
    hipHostFree(status_h);
    hipFree(keys_unique_d);
    hipFree(keys_in_d);
    hipFree(values_in_d);
    hipFree(offsets_out_d);
    hipFree(values_out_d);
    hipFree(status_d);
    hipFree(temp_d);

    hipDeviceSynchronize(); CUERR
}
