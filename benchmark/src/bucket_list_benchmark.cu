#include "hip/hip_runtime.h"
#include "common.cuh"
#include "warpcore.cuh"
#include "../../ext/hpc_helpers/include/io_helpers.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <set>
#include <tuple>
#include <vector>

template<class Key, class Value>
bool sufficient_memory(
    size_t key_store_capacity,
    size_t value_store_capacity,
    float headroom_factor = 1.1)
{
    const size_t key_handle_bytes = sizeof(Key)+sizeof(uint64_t);
    const size_t table_bytes = key_handle_bytes*key_store_capacity;
    const size_t value_bytes = std::min(sizeof(Value), sizeof(uint64_t));
    const size_t value_store_bytes = value_bytes * value_store_capacity;
    const size_t total_bytes = (table_bytes+value_store_bytes)*headroom_factor;

    size_t bytes_free, bytes_total;
    hipMemGetInfo(&bytes_free, &bytes_total); CUERR

    return (total_bytes <= bytes_free);
}

template<class HashTable>
HOSTQUALIFIER INLINEQUALIFIER
void bucket_list_benchmark(
    const typename HashTable::key_type * keys_d,
    const uint64_t max_keys,
    float key_load_factor,
    float value_load_factor,
    std::vector<uint64_t> input_sizes,
    std::vector<std::tuple<float, uint64_t, uint64_t>> slab_list_configs,
    typename HashTable::key_type seed = warpcore::defaults::seed<key_t>(),
    bool print_headers = true,
    uint8_t iters = 1,
    std::chrono::milliseconds thermal_backoff = std::chrono::milliseconds(100))
{
    using index_t = typename HashTable::index_type;
    using key_t = typename HashTable::key_type;
    using value_t = typename HashTable::value_type;

    const uint64_t max_unique_size = num_unique(keys_d, max_keys);

    std::cout << "unique_keys: " << max_unique_size << "\tvalues: " << max_keys << std::endl;

    const uint64_t key_store_capacity = max_unique_size / key_load_factor;
    const uint64_t value_store_capacity = max_keys / value_load_factor;

    key_t* unique_keys_d = nullptr;
    hipMalloc(&unique_keys_d, sizeof(key_t)*max_unique_size); CUERR
    value_t* values_d = nullptr;
    hipMalloc(&values_d, sizeof(value_t)*max_keys); CUERR
    index_t * offsets_d = nullptr;
    hipMalloc(&offsets_d, sizeof(index_t)*(max_keys+1)); CUERR

    const auto max_input_size =
        *std::max_element(input_sizes.begin(), input_sizes.end());

    if(max_input_size > max_keys)
    {
        std::cerr << "Maximum input size exceeded." << std::endl;
        exit(1);
    }

    if(!sufficient_memory<key_t, value_t>(key_store_capacity, value_store_capacity))
    {
        std::cerr << "Not enough GPU memory." << std::endl;
        exit(1);
    }

    for(const auto& size : input_sizes)
    {
        for(const auto& slab_list_config : slab_list_configs)
        {
            const float slab_grow_factor = std::get<0>(slab_list_config);
            const index_t min_slab_size = std::get<1>(slab_list_config);
            const index_t max_slab_size = std::get<2>(slab_list_config);

            HashTable hash_table(
                key_store_capacity,
                value_store_capacity,
                seed,
                slab_grow_factor,
                min_slab_size);

            Output<key_t,value_t> output;
            output.sample_size = size;
            output.key_capacity = hash_table.key_capacity();
            output.value_capacity = hash_table.value_capacity();

            output.insert_ms = benchmark_insert(
                hash_table, keys_d, values_d, size,
                iters, thermal_backoff);

            output.query_ms = benchmark_query_multi(
                hash_table, unique_keys_d, offsets_d, values_d,
                iters, thermal_backoff);

            output.key_load_factor = hash_table.key_load_factor();
            output.value_load_factor = hash_table.value_load_factor();
            output.density = hash_table.storage_density();
            output.relative_density = hash_table.relative_storage_density();
            output.status = hash_table.pop_status();

            std::cout << std::fixed
                << "grow_factor=" << slab_grow_factor
                << output.d << "min_slab_size=" << min_slab_size
                << output.d << "max_slab_size=" << max_slab_size
                << output.d;

            if(print_headers)
                output.print_with_headers();
            else
                output.print_without_headers();
        }
    }

    hipFree(unique_keys_d); CUERR
    hipFree(values_d); CUERR
    hipFree(offsets_d); CUERR
}

int main(int argc, char* argv[])
{
    using namespace warpcore;

    using key_t = std::uint32_t;
    using value_t = std::uint32_t;

    const uint64_t max_keys = 1UL << 27;

    uint64_t dev_id = 0;
    if(argc > 2) dev_id = std::atoi(argv[2]);
    hipSetDevice(dev_id); CUERR

    key_t * keys_d = nullptr;
    if(argc > 1)
        keys_d = load_keys<key_t>(argv[1], max_keys);
    else
        keys_d = generate_keys<key_t>(max_keys, 8);

    using hash_table_t = BucketListHashTable<
        key_t,
        value_t,
        defaults::empty_key<key_t>(),
        defaults::tombstone_key<key_t>(),
        storage::multi_value::BucketListStore<value_t, 29, 18, 15>,
        defaults::probing_scheme_t<key_t, 8>>;

    bucket_list_benchmark<hash_table_t>(
        keys_d, max_keys,
        0.90,
        0.50,
        {max_keys},
        {{1.1, 1, 0}},
        0x5ad0ded);

    hipFree(keys_d); CUERR
}
