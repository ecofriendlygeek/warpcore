#include <iostream>
#include <algorithm>
#include <numeric>
#include <vector>
#include <thread>
#include <chrono>
#include "warpcore.cuh"
#include "../../ext/hpc_helpers/include/io_helpers.h"

template<class Key, class Value>
bool sufficient_memory(size_t size, float load, float headroom_factor = 1.1)
{
    const size_t capacity = size/load;
    const size_t key_val_bytes = sizeof(Key)+sizeof(Value);
    const size_t table_bytes = key_val_bytes*capacity;
    const size_t io_bytes = key_val_bytes*size;
    const size_t total_bytes = (table_bytes+io_bytes)*headroom_factor;

    size_t bytes_free, bytes_total;
    hipMemGetInfo(&bytes_free, &bytes_total); CUERR

    return (total_bytes <= bytes_free);
}

uint64_t memory_partition(float factor = 0.4)
{
    size_t bytes_free, bytes_total;
    hipMemGetInfo(&bytes_free, &bytes_total); CUERR

    return bytes_free * factor;
}

template<class T>
uint64_t num_unique(const std::vector<T>& v) noexcept
{
    T * keys_d = nullptr;
    hipMalloc(&keys_d, sizeof(T) * v.size()); CUERR
    hipMemcpy(keys_d, v.data(), sizeof(T) * v.size(), H2D); CUERR

    auto set = warpcore::HashSet<T>(v.size());

    set.insert(keys_d, v.size());

    hipFree(keys_d);

    return set.size();
}

template<class HashTable>
HOSTQUALIFIER INLINEQUALIFIER
void multi_value_benchmark(
    const std::vector<typename HashTable::key_type>& keys,
    std::vector<uint64_t> input_sizes = {(1UL<<27)},
    std::vector<float> load_factors = {0.8},
    uint64_t dev_id = 0,
    bool print_headers = true,
    uint8_t iters = 5,
    std::chrono::milliseconds thermal_backoff = std::chrono::milliseconds(100))
{
    hipSetDevice(dev_id); CUERR

    using index_t = typename HashTable::index_type;
    using key_t = typename HashTable::key_type;
    using value_t = typename HashTable::value_type;

    const auto max_input_size =
        *std::max_element(input_sizes.begin(), input_sizes.end());
    const auto min_load_factor =
        *std::min_element(load_factors.begin(), load_factors.end());

    if(max_input_size > keys.size())
    {
        std::cerr << "Maximum input size exceeded." << std::endl;
        exit(1);
    }

    if(!sufficient_memory<key_t, value_t>(max_input_size, min_load_factor))
    {
        std::cerr << "Not enough GPU memory." << std::endl;
        exit(1);
    }

    key_t* keys_d = nullptr;
    hipMalloc(&keys_d, sizeof(key_t)*max_input_size); CUERR
    key_t* unique_keys_d = nullptr;
    hipMalloc(&unique_keys_d, sizeof(key_t)*max_input_size); CUERR
    value_t* values_d = nullptr;
    hipMalloc(&values_d, sizeof(value_t)*max_input_size); CUERR
    index_t * offsets_d = nullptr;
    hipMalloc(&offsets_d, sizeof(index_t)*(max_input_size+1)); CUERR

    hipMemcpy(keys_d, keys.data(), sizeof(key_t)*max_input_size, H2D); CUERR
    hipMemset(values_d, 1, sizeof(value_t)*max_input_size); CUERR

    for(auto size : input_sizes)
    {
        for(auto load : load_factors)
        {
            // const std::uint64_t capacity = float(size) / load;
            // const std::uint64_t capacity = float(size) / HashTable::bucket_size() / load;
            const float factor =
                float(sizeof(key_t) + sizeof(value_t)) /
                     (sizeof(key_t) + sizeof(value_t)*HashTable::bucket_size());

            const std::uint64_t capacity = size * factor / load;

            HashTable hash_table(capacity);

            std::vector<float> insert_times(iters);
            for(uint64_t i = 0; i < iters; i++)
            {
                hash_table.init();
                hipEvent_t insert_start, insert_stop;
                float t;
                hipEventCreate(&insert_start);
                hipEventCreate(&insert_stop);
                hipEventRecord(insert_start, 0);
                hash_table.insert(keys_d, values_d, size);
                hipEventRecord(insert_stop, 0);
                hipEventSynchronize(insert_stop);
                hipEventElapsedTime(&t, insert_start, insert_stop);
                hipDeviceSynchronize(); CUERR
                insert_times[i] = t;
                std::this_thread::sleep_for (thermal_backoff);
            }
            const float insert_time =
                *std::min_element(insert_times.begin(), insert_times.end());

            // std::cerr << "keys in table: " << hash_table.num_keys() << '\n';

            // auto key_set = hash_table.get_key_set();
            // std::cerr << "keys in set: " << key_set.size() << '\n';

            index_t key_size_out = 0;
            index_t value_size_out = 0;

            hash_table.retrieve_all_keys(unique_keys_d, key_size_out); CUERR

            std::vector<float> query_times(iters);
            for(uint64_t i = 0; i < iters; i++)
            {
                hipEvent_t query_start, query_stop;
                float t;
                hipEventCreate(&query_start);
                hipEventCreate(&query_stop);
                hipEventRecord(query_start, 0);
                hash_table.retrieve(
                    unique_keys_d,
                    key_size_out,
                    offsets_d,
                    offsets_d+1,
                    values_d,
                    value_size_out);
                hipEventRecord(query_stop, 0);
                hipEventSynchronize(query_stop);
                hipEventElapsedTime(&t, query_start, query_stop);
                hipDeviceSynchronize(); CUERR
                query_times[i] = t;
                std::this_thread::sleep_for(thermal_backoff);
            }
            const float query_time =
                *std::min_element(query_times.begin(), query_times.end());

            const uint64_t total_input_bytes = (sizeof(key_t) + sizeof(value_t))*size;
            uint64_t ips = size/(insert_time/1000);
            uint64_t qps = size/(query_time/1000);
            float itp = helpers::B2GB(total_input_bytes) / (insert_time/1000);
            float qtp = helpers::B2GB(total_input_bytes) / (query_time/1000);
            uint64_t key_capacity = hash_table.capacity();
            uint64_t value_capacity = hash_table.value_capacity();
            float key_load = hash_table.key_load_factor();
            float value_load = hash_table.value_load_factor();
            float density = hash_table.storage_density();
            float relative_density = hash_table.relative_storage_density();
            uint64_t table_bytes = hash_table.bytes_total();
            warpcore::Status status = hash_table.pop_status();

            if(print_headers)
            {
                const char d = ' ';

                std::cout << "N=" << size << std::fixed
                    << d << "key_capacity=" << key_capacity
                    << d << "value_capacity=" << value_capacity
                    << d << "bits_key=" << sizeof(key_t)*CHAR_BIT
                    << d << "bits_value=" << sizeof(value_t)*CHAR_BIT
                    << d << "mb_keys=" << uint64_t(helpers::B2MB(sizeof(key_t)*size))
                    << d << "mb_values=" << uint64_t(helpers::B2MB(sizeof(value_t)*size))
                    << d << "key_load=" << key_load
                    << d << "value_load=" << value_load
                    << d << "density=" << density
                    << d << "relative_density=" << relative_density
                    << d << "table_bytes=" << table_bytes
                    << d << "insert_ms=" << insert_time
                    << d << "query_ms=" << query_time
                    << d << "IPS=" << ips
                    << d << "QPS=" << qps
                    << d << "insert_GB/s=" << itp
                    << d << "query_GB/s=" << qtp
                    << d << "status=" << status << std::endl;
            }
            else
            {
                const char d = ' ';

                std::cout << std::fixed
                    << size
                    << d << capacity
                    << d << sizeof(key_t)*CHAR_BIT
                    << d << sizeof(value_t)*CHAR_BIT
                    << d << uint64_t(helpers::B2MB(sizeof(key_t)*size))
                    << d << uint64_t(helpers::B2MB(sizeof(value_t)*size))
                    << d << key_load
                    << d << value_load
                    << d << density
                    << d << relative_density
                    << d << table_bytes
                    << d << insert_time
                    << d << query_time
                    << d << ips
                    << d << qps
                    << d << itp
                    << d << qtp
                    << d << status << std::endl;
            }
        }
    }

    hipFree(keys_d); CUERR
    hipFree(values_d); CUERR
}

int main(int argc, char* argv[])
{
    using namespace warpcore;

    using key_t = std::uint32_t;
    using value_t = std::uint32_t;

    using mb1_hash_table_t = MultiBucketHashTable<
        key_t,
        value_t,
        defaults::empty_key<key_t>(),
        defaults::tombstone_key<key_t>(),
        defaults::empty_key<value_t>(),
        defaults::probing_scheme_t<key_t, 8>,
        storage::key_value::AoSStore<key_t, ArrayBucket<value_t,1>>>;

    using mb2_hash_table_t = MultiBucketHashTable<
        key_t,
        value_t,
        defaults::empty_key<key_t>(),
        defaults::tombstone_key<key_t>(),
        defaults::empty_key<value_t>(),
        defaults::probing_scheme_t<key_t, 8>,
        storage::key_value::AoSStore<key_t, ArrayBucket<value_t,2>>>;

    using mb4_hash_table_t = MultiBucketHashTable<
        key_t,
        value_t,
        defaults::empty_key<key_t>(),
        defaults::tombstone_key<key_t>(),
        defaults::empty_key<value_t>(),
        defaults::probing_scheme_t<key_t, 8>,
        storage::key_value::AoSStore<key_t, ArrayBucket<value_t,4>>>;

    using mb8_hash_table_t = MultiBucketHashTable<
        key_t,
        value_t,
        defaults::empty_key<key_t>(),
        defaults::tombstone_key<key_t>(),
        defaults::empty_key<value_t>(),
        defaults::probing_scheme_t<key_t, 8>,
        storage::key_value::AoSStore<key_t, ArrayBucket<value_t,8>>>;

    const uint64_t max_keys = 1UL << 27;
    uint64_t dev_id = 0;
    std::vector<key_t> keys;

    if(argc > 2) dev_id = std::atoi(argv[2]);

    if(argc > 1)
    {
        keys = helpers::load_binary<key_t>(argv[1], max_keys);
    }
    else
    {
        keys.resize(max_keys);

        key_t * keys_d = nullptr;
        hipMalloc(&keys_d, sizeof(key_t) * max_keys); CUERR

        helpers::lambda_kernel
        <<<SDIV(max_keys, 1024), 1024>>>
        ([=] DEVICEQUALIFIER
        {
            const uint64_t tid = blockDim.x * blockIdx.x + threadIdx.x;

            if(tid < max_keys)
            {
                keys_d[tid] = (tid % (max_keys / 8)) + 1;
            }
        });

        hipMemcpy(keys.data(), keys_d, sizeof(key_t) * max_keys, D2H); CUERR

        hipFree(keys_d); CUERR
    }

    multi_value_benchmark<mb1_hash_table_t>(
        keys,
        {max_keys},
        {0.8},
        dev_id);

    multi_value_benchmark<mb2_hash_table_t>(
        keys,
        {max_keys},
        {0.8},
        dev_id);

    multi_value_benchmark<mb4_hash_table_t>(
        keys,
        {max_keys},
        {0.8},
        dev_id);

    multi_value_benchmark<mb8_hash_table_t>(
        keys,
        {max_keys},
        {0.8},
        dev_id);
}
